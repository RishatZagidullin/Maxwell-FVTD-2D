#include "hip/hip_runtime.h"
#include "transport.cuh"

namespace advection
{

	__device__ double sigma_device(double const &r, double const &dr, int const &PML, int const & size)
	{
		if (r>(dr*(size-PML))) return pow((r-(size-PML)*dr)/(PML*dr),2)*3.0*log(10.0)*13.0/(PML*dr);
		else if (r < dr*(PML)) return pow((PML*dr-r)/(PML*dr),2)*3.0*log(10.0)*13.0/(PML*dr);
		else return 0;
	}

	void initialize_arrays(int solver_type, int size, bool*& is_boundary, double *& face_normals, int *& neighbor_ids, double *& tr_areas, double *& xbarys, double *& ybarys)
	{
		if (solver_type == 0)
		{
			is_boundary = new bool [size];
			neighbor_ids = new int [3*size];
			face_normals = new double [3*2*size];
			tr_areas = new double [size];
			xbarys =  new double [size];
			ybarys = new double [size];
		}
		else
		{
			hipMallocManaged((void **) &is_boundary, size*sizeof(bool));
			hipMallocManaged((void **) &neighbor_ids, 3*size*sizeof(int));
			hipMallocManaged((void **) &face_normals, size*3*2*sizeof(double));
			hipMallocManaged((void **) &tr_areas, size*sizeof(double));
			hipMallocManaged((void **) &xbarys, size*sizeof(double));
			hipMallocManaged((void **) &ybarys, size*sizeof(double));
		}
	}

	__global__ void solver_small_cuda(double *u, double dt, double *velocities, double *interpolated_velocities, bool * is_boundary, int * neighbor_ids, double * face_normals, double * tr_areas, double * ybarys, double * xbarys, int size, bool if_y, bool if_h, double t);	

	void dealloc_arrays(int solver_type, bool*& is_boundary, double *& face_normals, int *& neighbor_ids, double *& tr_areas, double *& xbarys, double *& ybarys)
	{
		if (solver_type == 0)
		{
			delete [] face_normals;
			delete [] neighbor_ids;
			delete [] tr_areas;
			delete [] xbarys;
			delete [] ybarys;
			delete [] is_boundary;
		}
		else
		{
			hipFree(face_normals);
			hipFree(neighbor_ids);
			hipFree(tr_areas);
			hipFree(is_boundary);
			hipFree(xbarys);
			hipFree(ybarys);
			hipFree(is_boundary);
		}
	}
	void call_solver_small_cuda(double *&u, double const dt, double *&velocities, bool * is_boundary, double * face_normals, int * neighbor_ids, double * tr_areas, double * xbarys, double * ybarys, int size, bool if_y, bool if_h, double t)
	{
		double * interpolated_velocities;
		hipMallocManaged((void **) &interpolated_velocities, size*3*2*sizeof(double));
		dim3 block(32);
		dim3 grid((size+block.x-1)/block.x);
		solver_small_cuda<<<grid, block>>>(u, dt, velocities, interpolated_velocities, is_boundary, neighbor_ids, face_normals, tr_areas, ybarys, xbarys, size, if_y, if_h, t);
		hipDeviceSynchronize();
		hipFree(interpolated_velocities);			
	}
	
	__global__ void solver_small_cuda(double *u, double dt, double *velocities, double *interpolated_velocities, bool * is_boundary, int * neighbor_ids, double * face_normals, double * tr_areas, double * ybarys, double * xbarys, int size, bool if_y, bool if_h, double t)
	{
		int numThreads = blockDim.x * blockDim.y * gridDim.x * gridDim.y;
		int global_id = (threadIdx.y + blockIdx.y * blockDim.y)*blockDim.x*gridDim.x + (threadIdx.x + blockIdx.x * blockDim.x);
		for (int j = global_id; j < size; j+=numThreads)
		{	
			for (int k = 0; k < 3; k++)
			{
				if (neighbor_ids[j*3+k] != -1)
				{
					if (if_h && if_y)
					{
						if (is_boundary[j]!=is_boundary[neighbor_ids[j*3+k]] && ybarys[j] < 0.6 && ybarys[j] > 0.1 && ybarys[neighbor_ids[j*3+k]] < 0.6 && ybarys[neighbor_ids[j*3+k]] > 0.1)
						{
							interpolated_velocities[2*(j*3+k)] = (is_boundary[j] ? 1.0 : -1.0)*cos(25.0*(xbarys[neighbor_ids[j*3+k]] - 1.5*t))+velocities[2*j]+velocities[2*neighbor_ids[j*3+k]];
							interpolated_velocities[2*(j*3+k)+1] = velocities[2*j+1]+velocities[2*neighbor_ids[j*3+k]+1];
						}
						else
						{
							interpolated_velocities[2*(j*3+k)] = velocities[2*neighbor_ids[j*3+k]] + velocities[2*j];
							interpolated_velocities[2*(j*3+k)+1] = velocities[2*neighbor_ids[j*3+k]+1] + velocities[2*j+1];
						}
					}
					else
					{
						interpolated_velocities[2*(j*3+k)] = velocities[2*neighbor_ids[j*3+k]] + velocities[2*j];
						interpolated_velocities[2*(j*3+k)+1] = velocities[2*neighbor_ids[j*3+k]+1] + velocities[2*j+1];
					}
				}
				else
				{
					interpolated_velocities[2*(j*3+k)] = velocities[2*j]+0.0;
					interpolated_velocities[2*(j*3+k)+1] = velocities[2*j+1]+0.0;
				}
			}
			double temp = 0.0;
			for (int k = 0; k < 3; k++)
			{
				temp += interpolated_velocities[2*(3*j+k)] * face_normals[2*(j*3+k)] + interpolated_velocities[2*(3*j+k)+1] * face_normals[2*(j*3+k)+1]; 
			}
			if (!if_h) u[j] = u[j] - dt * (temp/ tr_areas[j] + (if_y ? 0.5*pow(2.25, 0.5)*sigma_device(ybarys[j], 0.01, 7, 70)*u[j] : 0.5*pow(2.25, 0.5)*sigma_device(xbarys[j], 0.01, 15, 150)*u[j]))/(1.0+0.5*dt*( if_y ? pow(2.25, 0.5)*sigma_device(ybarys[j], 0.01, 7, 70) : pow(2.25, 0.5)*sigma_device(xbarys[j], 0.01, 15, 150)));
			else u[j] = u[j] - dt * (temp / tr_areas[j] + 0.5*pow(1.0/2.25, 0.5)*sigma_device(ybarys[j], 0.01, 7, 70)*u[j] + 0.5*pow(1.0/2.25, 0.5)*sigma_device(xbarys[j], 0.01, 15, 150)*u[j])/(1.0+0.5*dt*( pow(1.0/2.25, 0.5)*sigma_device(ybarys[j], 0.01, 7, 70) + pow(1.0/2.25, 0.5)*sigma_device(xbarys[j], 0.01, 15, 150)));
		}
	}
	
//mu_1 = 1.0, mu_2 = 1.0, epsilon_1 = 1.0, epsilon_2 = 2.25
}
